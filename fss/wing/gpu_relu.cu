#include "hip/hip_runtime.h"
// Author: Neha Jawalkar
// Copyright:
// 
// Copyright (c) 2024 Microsoft Research
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "gpu_relu.h"
#include "utils/gpu_comms.h"

namespace wing
{

    template <typename T>
    __global__ void genSelectExtKernel(T* inputMask, T* outputMask, u8* rs, T* re, T* v, T* p, T* q, int bin, int bout, int N){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if(i < N){
            assert(rs[i] == 0 || rs[i] == 1);
            auto rmsb = gpuMsb(inputMask[i], bin);
            v[i] = (1 - rs[i]) * inputMask[i] - outputMask[i];
            gpuMod(v[i], bout);
            p[i] = rs[i] * rmsb;
            q[i] = (1 - rs[i]) * rmsb;
            re[i] = inputMask[i] - outputMask[i] - outputMask[i];
            gpuMod(re[i], bout);
        }
    }

    template <typename T>
    T* gpuKeyGenSelectExt(uint8_t** key_as_bytes, int party, int bin, int bout, int N, u8* rs, T* inputMask){
        auto outputMask = randomGEOnGpu<T>(N, bout);
        T* re = (T*)gpuMalloc(N * sizeof(T));
        T* v = (T*)gpuMalloc(N * sizeof(T));
        T* p = (T*)gpuMalloc(N * sizeof(T));
        T* q = (T*)gpuMalloc(N * sizeof(T));
        
        genSelectExtKernel<<<(N - 1) / 256 + 1, 256>>>(inputMask, outputMask, rs, re, v, p, q, bin, bout, N);
        writeShares<T, T>(key_as_bytes, party, N, re, bout);
        writeShares<u8, T>(key_as_bytes, party, N, rs, bout);
        writeShares<T, T>(key_as_bytes, party, N, v, bout);
        writeShares<T, T>(key_as_bytes, party, N, p, bout);
        writeShares<T, T>(key_as_bytes, party, N, q, bout);
        
        gpuFree(inputMask);
        gpuFree(v);
        gpuFree(p);
        gpuFree(q);
        return outputMask;
    }

    // need to check this
    // drelu mask is used as input mask for the next set of protocols
    // do we need something better than u64?
    template <typename T>
    u8 *keygenDRelu(uint8_t **key_as_bytes, int party, int bin, int N, T *d_rin, AESGlobalContext *gaes)
    {
        // need to write everything in the proper format
        // printf("%d, %d\n", bin, N);
        gpuKeyGenDCF<T>(key_as_bytes, party, bin, 1, N, d_rin, T(1), gaes);
        auto d_dreluMask = randomGEOnGpu<u8>(N, 1);
        writeShares<u8, u8>(key_as_bytes, party, N, d_dreluMask, 1);
        return d_dreluMask;
    }
    // need to check this
    template <typename T>
    std::pair<u8 *, T *> gpuGenTwoRoundReluKey(uint8_t **key_as_bytes, int party, int bin, int bout, int N, T *d_inputMask, AESGlobalContext *gaes)
    {
        writeInt(key_as_bytes, bin);
        writeInt(key_as_bytes, bout);
        writeInt(key_as_bytes, N);
        auto d_dreluMask = keygenDRelu(key_as_bytes, party, bin, N, d_inputMask, gaes);
        auto d_outputMask = gpuKeyGenSelect<T, T, u8>(key_as_bytes, party, N, d_inputMask, d_dreluMask, bout);
        return std::make_pair(d_dreluMask, d_outputMask);
    }

    template <typename T>
    std::pair<u32 *, T *> gpuTwoRoundRelu(SigmaPeer *peer, int party, GPU2RoundReLUKey<T> k, T *d_I, AESGlobalContext *gaes, Stats *s)
    {
        std::vector<u32 *> h_dreluMask = {k.dreluKey.dReluMask};
        auto d_drelu = gpuDcf<T, 2, dReluPrologue, dReluEpilogue<false>>(k.dreluKey.dcfKey, party, d_I, gaes, s, &h_dreluMask);
        peer->reconstructInPlace(d_drelu, 1, k.N, s);
        auto d_relu = gpuSelect<T, T, 0, 0>(peer, party, k.bout, k.selectKey, (u32 *)d_drelu, d_I, s, true);
        return std::make_pair(d_drelu, d_relu);
    }

    template <typename T>
    __global__ void reluExtendMuxKernel(int party, int bin, /*int f,*/ int N, T *x, T* y, T *oneHot, T *outMask, u32 *drelu, u32 *xLTRin)
    {
        int j = blockIdx.x * blockDim.x + threadIdx.x;
        if (j < N)
        {
            int posInBlock = threadIdx.x & 0xf;
            u32 d = (((u32 *)drelu)[j / 16] >> (2 * posInBlock)) & 3;
            u32 w = (((u32 *)xLTRin)[j / 16] >> (2 * posInBlock)) & 3;
            u32 i = (2 * d + w) & 3;
            // should i store this table transposed instead?
            // will always access sequential elements so might benefit from locality within a thread
            T rotatedP3 = oneHot[4 * j + ((2 - i) & 3)];
            T rotatedP4 = oneHot[4 * j + ((3 - i) & 3)];
            T xIn = x[j];

            y[j] = xIn * rotatedP3 + (xIn + (1ULL << (bin))) * rotatedP4 + outMask[2 * j + (d & 1)];
            u64 dreluBit = static_cast<u64>(d & 1);
            writePackedOp(xLTRin, dreluBit, 1, N);
        }
    }

    template <typename T>
    T* gpuReluExtendMux(int party, int bin, int N,
                              T *d_I, T *h_oneHot, T *h_outMask, u32 *d_drelu,
                              u32 *d_xLTRin, Stats *s)
    {
        auto d_out = (T*) gpuMalloc(N * sizeof(T));
        auto d_oneHot = (T *)moveToGPU((uint8_t *)h_oneHot, 4 * N * sizeof(T), s);
        auto d_outMask = (T *)moveToGPU((uint8_t *)h_outMask, 2 * N * sizeof(T), s);
        reluExtendMuxKernel<<<(N - 1) / 128 + 1, 128>>>(party, bin, N, d_I, d_out, d_oneHot, d_outMask, d_drelu, d_xLTRin);
        checkCudaErrors(hipDeviceSynchronize());
        gpuFree(d_oneHot);
        gpuFree(d_outMask);
        return d_out;
    }

    template <typename T>
    __global__ void reluExtendMuxKeyKernel(int bin, int bout, int N, T *d_inputMask, u8 *d_dreluMask, u8 *d_dcfMask, T *d_randomMask, T *d_oneHot, T *d_outMask)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < N)
        {
            auto onePos = (-(2 * d_dreluMask[i] + d_dcfMask[i])) & T(3);
            assert(onePos < 4);
            for (int j = 0; j < 4; j++)
            {
                d_oneHot[4 * i + j] = (j == onePos ? T(1) : T(0));
            }
            int outputMask0Idx = d_dreluMask[i] & T(1);
            int outputMask1Idx = 1 - outputMask0Idx;
            d_outMask[2 * i + outputMask0Idx] = d_randomMask[i];
            d_outMask[2 * i + outputMask1Idx] = d_randomMask[i] - d_inputMask[i];
            d_dreluMask[i] &= T(1);
        }
    }

    template <typename T>
    T *genReluExtendMuxKey(uint8_t **key_as_bytes, int party, int bin, int bout, int N, T *d_inputMask, u8 *d_dreluMask, u8 *d_dcfMask)
    {
        auto d_randomMask = randomGEOnGpu<T>(N, bout);
        auto d_oneHot = (T *)gpuMalloc(4 * N * sizeof(T));
        auto d_outMask = (T *)gpuMalloc(2 * N * sizeof(T));
        reluExtendMuxKeyKernel<<<(N - 1) / 256 + 1, 256>>>(bin, bout, N, d_inputMask, d_dreluMask, d_dcfMask, d_randomMask, d_oneHot, d_outMask);
        writeShares<T, T>(key_as_bytes, party, 4 * N, d_oneHot, bout);
        writeShares<T, T>(key_as_bytes, party, 2 * N, d_outMask, bout);
        gpuFree(d_oneHot);
        gpuFree(d_outMask);
        return d_randomMask;
    }

    template <typename T>
    std::pair<u8*, T*> gpuKeyGenReluZeroExt(uint8_t **key_as_bytes, int party, int bin, int bout, int N, T *d_inputMask, AESGlobalContext* g)
    {
        writeInt(key_as_bytes, bin);
        writeInt(key_as_bytes, bout);
        writeInt(key_as_bytes, N);
        auto cur_bytes = *key_as_bytes;
        auto d_dReluMask = dpf::gpuKeyGenDRelu(key_as_bytes, party, bin, N, d_inputMask, g);
        int key_as_bytes_sz = *key_as_bytes - cur_bytes;
        printf("DRelu Key size=%d\n", key_as_bytes_sz);
        cur_bytes = *key_as_bytes;
        auto d_outputMask = gpuKeyGenSelectExt(key_as_bytes, party, bin, bout, N, d_dReluMask, d_inputMask);
        key_as_bytes_sz = *key_as_bytes - cur_bytes;
        printf("Select Key size=%d\n", key_as_bytes_sz);
        return std::make_pair(d_dReluMask, d_outputMask);
    }

    template <typename T>
    std::pair<u8 *, T *> gpuKeygenReluExtend(uint8_t **key_as_bytes, int party, int bin, int bout, int N, T *d_inputMask, AESGlobalContext* g)
    {
        writeInt(key_as_bytes, bin);
        writeInt(key_as_bytes, bout);
        writeInt(key_as_bytes, N);
        gpuKeyGenDCF(key_as_bytes, party, bin, 2, N, d_inputMask, T(1), g);
        auto d_dreluMask = randomGEOnGpu<u8>(N, 2);
        // checkCudaErrors(hipMemset(d_dreluMask, 0, N));
        auto d_dcfMask = randomGEOnGpu<u8>(N, 2);
        // checkCudaErrors(hipMemset(d_dcfMask, 0, N));
        writeShares<u8, u8>(key_as_bytes, party, N, d_dreluMask, 2);
        writeShares<u8, u8>(key_as_bytes, party, N, d_dcfMask, 2);
        auto d_randomMask = genReluExtendMuxKey(key_as_bytes, party, bin, bout, N, d_inputMask, d_dreluMask, d_dcfMask);
        // gpuFree(d_inputMask);
        gpuFree(d_dcfMask);
        // gpuFree(d_dreluMask);
        return std::make_pair(d_dreluMask, d_randomMask);
    }

    template <typename T>
    std::pair<u32 *, T *> gpuReluExtend(SigmaPeer *peer, int party, GPUReluExtendKey<T> k, T *d_I, AESGlobalContext *g, Stats *s)
    {
        std::vector<u32 *> h_masks = {k.dReluKey.dReluMask, k.dcfMask};
        auto d_dcf = gpuDcf<T, 2, dReluPrologue, dReluEpilogue<true>>(k.dReluKey.dcfKey, party, d_I, g, s, &h_masks);
        peer->reconstructInPlace(d_dcf, 2, 2 * k.dReluKey.dcfKey.memSzOut * 4, s);
        auto d_drelu = d_dcf;
        auto d_xLTRin = (u32 *)(((u8 *)d_dcf) + k.dReluKey.dcfKey.memSzOut);
        auto d_relu = gpuReluExtendMux(party, k.bin, k.N, d_I, k.oneHot, k.outMask, d_drelu, d_xLTRin, s);
        peer->reconstructInPlace(d_relu, k.bout, k.N, s);
        return std::make_pair(d_drelu, d_relu);
    }
    
    template <typename T>
    __global__ void ReluZeroExtMuxKernel(int party, int bin, int bout, int N, T* d_I, u32* d_dcf, T* d_re, T* d_rs, T* d_v, T* d_p, T* d_q, T* res)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < N)
        {
            d_I[i] = d_I[i] + (1ULL << (bin - 2));
            gpuMod(d_I[i], bin);
            auto t = (1ULL - gpuMsb(d_I[i], bin)) * (1ULL << bin);
            int laneId = threadIdx.x & 0x1f;
            auto dhat = ((d_dcf[i / 32] >> laneId) & 1ULL);
            d_I[i] = d_I[i] - (1ULL << (bin - 2));
            gpuMod(d_I[i], bout);
            assert(dhat == 0 || dhat == 1);
            
            if(dhat){
                res[i] = (T(party) - d_rs[i]) * d_I[i] + t * (d_q[i]) - d_v[i];
            }
            else{
                res[i] = d_rs[i] * d_I[i] + t * d_p[i] + d_v[i] - d_re[i];
            }
            gpuMod(res[i], bout);
        }
    }

    template <typename T>
    T* gpuReluZeroExtMux(int party, int bin, int bout, int N, GPUSelectExtKey<T> k, T* d_I, u32* d_dcf, Stats *s){
        auto d_relu = (T*)gpuMalloc(N * sizeof(T));
        auto d_re = (T *)moveToGPU((uint8_t *)k.re, N * sizeof(T), s);
        auto d_rs = (T *)moveToGPU((uint8_t *)k.rs, N * sizeof(T), s);
        auto d_v = (T *)moveToGPU((uint8_t *)k.v, N * sizeof(T), s);
        auto d_p = (T *)moveToGPU((uint8_t *)k.p, N * sizeof(T), s);
        auto d_q = (T *)moveToGPU((uint8_t *)k.q, N * sizeof(T), s);
        ReluZeroExtMuxKernel<<<(N - 1) / 256 + 1, 256>>>(party, bin, bout, N, d_I, d_dcf, d_re, d_rs, d_v, d_p, d_q, d_relu);
        gpuFree(d_re);
        gpuFree(d_rs);
        gpuFree(d_v);
        gpuFree(d_p);
        gpuFree(d_q);
        return d_relu;
    }

    template <typename T>
    std::pair<u32 *, T *> gpuReluZeroExt(SigmaPeer *peer, int party, GPUReluZeroExtKey<T> k, T *d_I, AESGlobalContext *g, Stats *s, bool reconstruct=true)
    {
        std::vector<u32 *> h_mask({k.dReluKey.mask});
        auto d_dcf = dpf::gpuDcf<T, 1, dpf::dReluPrologue<0>, dpf::dReluEpilogue<0, false>>(k.dReluKey.dpfKey, party, d_I, g, s, &h_mask);
        peer->reconstructInPlace(d_dcf, 1, k.N, s); 
        auto d_relu = gpuReluZeroExtMux(party, k.bin, k.bout, k.N, k.selectKey, d_I, d_dcf, s);
        if (reconstruct)
            peer->reconstructInPlace(d_relu, k.bout, k.N, s);
        return std::make_pair(d_dcf, d_relu);
    }
}