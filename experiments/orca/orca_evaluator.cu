// 
// Copyright:
// 
// Copyright (c) 2024 Microsoft Research
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <cassert>
#include <cstddef>
#include <cstdint>
#include <chrono>
#include <cstdio>
#include <fcntl.h>
#include <filesystem>
#include <omp.h>
#include <string>

#include "utils/gpu_data_types.h"
#include "utils/gpu_file_utils.h"
#include "utils/misc_utils.h"
#include "utils/gpu_comms.h"
#include "utils/gpu_mem.h"
#include "utils/hip/hip_runtime_api.h"
#include "utils/gpu_random.h"
#include "../datasets/gpu_data.h"

#include "nn/orca/gpu_layer.h"
#include "nn/orca/gpu_model.h"

#include "cnn.h"
#include "model_accuracy.h"

#include <sytorch/softmax.h>
#include <sytorch/backend/llama_base.h>

#include "hip/hip_runtime_api.h"
#include "utils/wan_config.h"

u64 *gpuSoftmax(int batchSz, int numClasses, int party, SigmaPeer *peer, u64 *d_I, u64 *labels, bool secfloat, LlamaBase<u64> *llama)
{
    Tensor4D<u64> inp(batchSz, numClasses, 1, 1);
    Tensor4D<u64> softmaxOp(batchSz, numClasses, 1, 1);

    size_t memSz = batchSz * numClasses * sizeof(u64);
    moveIntoCPUMem((u8 *)inp.data, (u8 *)d_I, memSz, NULL);
    gpuFree(d_I);
    if (secfloat)
    {
        softmax_secfloat(inp, softmaxOp, dcf::orca::global::scale, LlamaConfig::party);
    }
    else
    {
        pirhana_softmax(inp, softmaxOp, dcf::orca::global::scale);
        // softmax<u64,dcf::orca::global::scale>(inp, softmaxOp);
    }
    for (int img = 0; img < batchSz; img++)
    {
        for (int c = 0; c < numClasses; c++)
        {
            softmaxOp(img, c, 0, 0) -= (labels[numClasses * img + c] * (((1LL << dcf::orca::global::scale)) / batchSz));
        }
    }
    reconstruct(inp.d1 * inp.d2, softmaxOp.data, 64);
    d_I = (u64 *)moveToGPU((u8 *)softmaxOp.data, memSz, NULL);
    return d_I;
}

void trainModel(dcf::orca::GPUModel<u64> *m, u8 **keyBuf, int party, SigmaPeer *peer, u64 *data, u64 *labels, AESGlobalContext *g, bool secfloat, LlamaBase<u64> *llama, int epoch, int iteration)
{
    auto start = std::chrono::high_resolution_clock::now();
    size_t inpMemSz = m->inpSz * sizeof(u64);
    auto d_I = (u64 *)moveToGPU((u8 *)data, inpMemSz, &(m->layers[0]->s));
    u64 *d_O;
    for (int i = 0; i < m->layers.size(); i++)
    {
        // std::cout << "Read Key Layer " << i << " begin" << std::endl;
        m->layers[i]->readForwardKey(keyBuf);
        // std::cout << "Layer " << i << " begin" << std::endl;
        d_O = m->layers[i]->forward(peer, party, d_I, g);
        // std::cout << "Layer " << i << " done" << std::endl;
        if (d_O != d_I)
            gpuFree(d_I);
        d_I = d_O;
    }
    checkCudaErrors(hipDeviceSynchronize());
    d_I = gpuSoftmax(m->batchSz, m->classes, party, peer, d_I, labels, secfloat, llama);
    // std::cout << "Softmax finished" << std::endl;
    for (int i = m->layers.size() - 1; i >= 0; i--)
    {
        m->layers[i]->readBackwardKey(keyBuf, epoch);
        d_I = m->layers[i]->backward(peer, party, d_I, g, epoch);
        // std::cout << "Layer " << i << " backward done" << std::endl;
    }
}

void trainModelPerf(dcf::orca::GPUModel<u64> *m, u8 **keyBuf, int party, SigmaPeer *peer, u64 *data, u64 *labels, AESGlobalContext *g, bool secfloat, LlamaBase<u64> *llama, int epoch, int iteration, int & float_softmax_time)
{
    auto start = std::chrono::high_resolution_clock::now();
    size_t inpMemSz = m->inpSz * sizeof(u64);
    auto d_I = (u64 *)moveToGPU((u8 *)data, inpMemSz, &(m->layers[0]->s));
    u64 *d_O;
    for (int i = 0; i < m->layers.size(); i++)
    {
        // std::cout << "Read Key Layer " << i << " begin" << std::endl;
        m->layers[i]->readForwardKey(keyBuf);
        // std::cout << "Layer " << i << " begin" << std::endl;
        d_O = m->layers[i]->forward(peer, party, d_I, g);
        // std::cout << "Layer " << i << " done" << std::endl;
        if (d_O != d_I)
            gpuFree(d_I);
        d_I = d_O;
    }
    checkCudaErrors(hipDeviceSynchronize());
    auto computeStart = std::chrono::high_resolution_clock::now();
    d_I = gpuSoftmax(m->batchSz, m->classes, party, peer, d_I, labels, secfloat, llama);
    auto computeEnd = std::chrono::high_resolution_clock::now();
    float_softmax_time += std::chrono::duration_cast<std::chrono::milliseconds>(computeEnd - computeStart).count();
    // std::cout << "Softmax finished" << std::endl;
    for (int i = m->layers.size() - 1; i >= 0; i--)
    {
        m->layers[i]->readBackwardKey(keyBuf, epoch);
        d_I = m->layers[i]->backward(peer, party, d_I, g, epoch);
        // std::cout << "Layer " << i << " backward done" << std::endl;
    }
}

u64 getKeySz(std::string dir, std::string modelName)
{
    std::ifstream kFile(dir + modelName + ".txt");
    u64 keySz;
    kFile >> keySz;
    return keySz;
}

void rmWeights(std::string lossDir, int party, int l, int k)
{
    assert(std::filesystem::remove(lossDir + "weights_mask_" + std::to_string(party) + "_" + std::to_string(l) + "_" + std::to_string(k) + ".dat"));
    assert(std::filesystem::remove(lossDir + "masked_weights_" + std::to_string(party) + "_" + std::to_string(l) + "_" + std::to_string(k) + ".dat"));
}

void evaluatorE2E(std::string modelName, std::string dataset, int party, std::string ip, std::string weightsFile, bool floatWeights, int epochs, int blocks, int blockSz, int batchSz, int H, int W, int C, bool secfloat, bool momentum, std::string keyDir, bool fake_offline = true)
{
    AESGlobalContext g;
    initAESContext(&g);
    initGPUMemPool();
    initGPURandomness();
    initCPURandomness();
    // assert(epochs < 6);

    omp_set_num_threads(2);

    printf("Sync=%d\n", sync);
    printf("Opening fifos\n");
    char one = 1;
    char two = 2;

    auto trainingDir = "output/P" + std::to_string(party) + "/training/";
    auto expName = modelName + "-" + std::to_string(epochs) + "e-" + std::to_string(blocks) + "b";
    auto lossDir = trainingDir + "loss/" + expName + "/";
    auto weightsDir = lossDir + "weights/";
    auto keySzDir = trainingDir + "keysize/";
    std::ofstream lossFile(lossDir + "loss.txt");
    std::ofstream accFile(lossDir + "accuracy.txt");

    dcf::orca::GPUModel<u64> *m = getGPUModel<u64>(modelName, Tensor<u64>(nullptr, {(u64)batchSz, (u64)H, (u64)W, (u64)C}));
    m->setTrain(momentum);
    printf("Model created\n");
    m->initWeights(weightsFile, floatWeights);
    printf("Weights initialized\n");

    u8 *keyBuf1, *keyBuf2, *curKeyBuf, *nextKeyBuf;
    u64 keySz = getKeySz(keySzDir, modelName);
    getAlignedBuf(&keyBuf1, keySz);
    getAlignedBuf(&keyBuf2, keySz);
    int curBuf = 0;
    curKeyBuf = keyBuf1;
    nextKeyBuf = keyBuf2;

    SigmaPeer *peer = new GpuPeer(true);
    LlamaBase<u64> *llama = nullptr;

    // automatically truncates by scale
    LlamaConfig::party = party + 2;
    LlamaConfig::bitlength = dcf::orca::global::bw;
    llama = new LlamaBase<u64>();
    if (LlamaConfig::party == SERVER)
        llama->initServer(ip, (char **)&curKeyBuf);
    else
        llama->initClient(ip, (char **)&curKeyBuf);
    peer->peer = LlamaConfig::peer;

    if (secfloat)
        secfloat_init(party + 1, ip);
    
    std::string keyFile = keyDir + modelName + "_training_key" + std::to_string(party);
    // dropOSPageCache();
    std::chrono::duration<int64_t, std::milli> onlineTime = std::chrono::duration<int64_t, std::milli>::zero();
    std::chrono::duration<int64_t, std::milli> computeTime = std::chrono::duration<int64_t, std::milli>::zero();
    uint64_t keyReadTime = 0;
    size_t commBytes = 0;
    printf("Starting training\n");
    
    Dataset d = readDataset(dataset, party);
    int fd = openForReading(keyFile + "_" + to_string(0) + "_" + to_string(0) + "_" + std::to_string(0) + ".dat");
    for (int l = 0; l < epochs; l++)
    {
        for (int k = 0; k < blocks; k++)
        {
            initGPUMemPool();
            // Open the key file for reading
            // uncomment for end to end run
            peer->sync();
            auto startComm = peer->bytesReceived();
            auto start = std::chrono::high_resolution_clock::now();
            for (int j = 0; j < blockSz; j++)
            {
                readKey(fd, keySz, curKeyBuf, &keyReadTime);
                peer->sync();
                auto computeStart = std::chrono::high_resolution_clock::now();
                auto labelsIdx = (k * blockSz + j) * batchSz * d.classes;
                int dataIdx = (k * blockSz + j) * d.H * d.W * d.C * batchSz;
                trainModel(m, &curKeyBuf, party, peer, &(d.data[dataIdx]), &(d.labels[labelsIdx]), &g, secfloat, llama, l, l * blocks * blockSz + k * blockSz + j);
                auto computeEnd = std::chrono::high_resolution_clock::now();
                auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(computeEnd - computeStart);
                computeTime += elapsed;  
                curKeyBuf = &keyBuf1[0]; 
            }
            auto end = std::chrono::high_resolution_clock::now();
            auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
            onlineTime += elapsed;
            printf("Online time (ms): %lu\n", elapsed.count());
            auto endComm = peer->bytesReceived();
            commBytes += (endComm - startComm);
            std::pair<double, double> res;
            m->dumpWeights(weightsDir + "masked_weights_" + std::to_string(party) + "_" + std::to_string(l) + "_" + std::to_string(k) + "_" + std::to_string(blockSz-1) + ".dat");
            if (dataset == "mnist")
            {
                std::cout << "Getting loss for MNIST" << std::endl;
                res = getLossMNIST<i64>(modelName, (u64)dcf::orca::global::scale, weightsDir, party, l, k, blockSz-1, true);
            }
            else
            {
                std::cout << "Getting loss for CIFAR10" << std::endl;
                res = getLossCIFAR10<i64>(modelName, (u64)dcf::orca::global::scale, weightsDir, party, l, k, blockSz-1, true);
            }
            auto accuracy = res.first;
            auto loss = res.second;
            printf("Accuracy=%lf, Loss=%lf\n", accuracy, loss);
            lossFile << loss << std::endl;
            accFile << accuracy << std::endl;   
        }
    }
    close(fd);


    LlamaConfig::peer->close();
    int iterations = epochs * blocks * blockSz;
    commBytes += secFloatComm;
    std::ofstream stats(trainingDir + expName + ".txt");
    auto statsString = "Total time taken (ms): " + std::to_string(onlineTime.count()) + "\nTotal bytes communicated: " + std::to_string(commBytes) + "\nSecfloat softmax bytes: " + std::to_string(inputOnlineComm + secFloatComm);

    auto avgKeyReadTime = (double)keyReadTime / (double)iterations;
    auto avgComputeTime = (double)computeTime.count() / (double)iterations;

    double commPerIt = (double)commBytes / (double)iterations;
    statsString += "\nAvg key read time (ms): " + std::to_string(avgKeyReadTime) + "\nAvg compute time (ms): " + std::to_string(avgComputeTime);
    statsString += "\nComm per iteration (B): " + std::to_string(commPerIt);
    stats << statsString;
    stats.close();
    std::cout << statsString << std::endl;
    lossFile.close();
    accFile.close();
    destroyCPURandomness();
    destroyGPURandomness();
}

void evaluatorE2EFakeOffline(std::string modelName, std::string dataset, int party, std::string ip, std::string weightsFile, bool floatWeights, int epochs, int blocks, int blockSz, int batchSz, int H, int W, int C, bool secfloat, bool momentum, std::string keyDir, int device = 0)
{
    WanParameter wanParams;
    AESGlobalContext g;
    initAESContext(&g);    
    initGPURandomness();
    initCPURandomness();
    initGPUMemPool();
    // assert(epochs < 6);

    omp_set_num_threads(2);

    printf("Sync=%d\n", sync);
    printf("Opening fifos\n");
    char one = 1;
    char two = 2;

    auto trainingDir = "output/P" + std::to_string(party) + "/training/";
    auto expName = modelName + "-" + std::to_string(epochs) + "e-" + std::to_string(blocks) + "b";
    auto lossDir = trainingDir + "loss/" + expName + "/";
    auto weightsDir = lossDir + "weights/";
    auto keySzDir = trainingDir + "keysize/";
    std::ofstream lossFile(lossDir + "loss.txt");
    std::ofstream accFile(lossDir + "accuracy.txt");

    dcf::orca::GPUModel<u64> *m = getGPUModel<u64>(modelName, Tensor<u64>(nullptr, {(u64)batchSz, (u64)H, (u64)W, (u64)C}));
    m->setTrain(momentum);
    printf("Model created\n");
    m->initWeights(weightsFile, floatWeights);
    printf("Weights initialized\n");

    u8 *keyBuf1, *keyBuf2, *curKeyBuf, *nextKeyBuf;
    u64 keySz = getKeySz(keySzDir, modelName);
    getAlignedBuf(&keyBuf1, keySz);
    getAlignedBuf(&keyBuf2, keySz);
    int curBuf = 0;
    curKeyBuf = keyBuf1;
    nextKeyBuf = keyBuf2;

    SigmaPeer *peer = new GpuPeer(true);
    LlamaBase<u64> *llama = nullptr;

    // automatically truncates by scale
    LlamaConfig::party = party + 2;
    LlamaConfig::bitlength = dcf::orca::global::bw;
    llama = new LlamaBase<u64>();
    if (LlamaConfig::party == SERVER)
        llama->initServer(ip, (char **)&curKeyBuf);
    else
        llama->initClient(ip, (char **)&curKeyBuf);
    peer->peer = LlamaConfig::peer;

    if (secfloat)
        secfloat_init(party + 1, ip);
    
    std::string keyFile = keyDir + modelName + "_training_key" + std::to_string(party);
    // dropOSPageCache();
    std::chrono::duration<int64_t, std::milli> onlineTime = std::chrono::duration<int64_t, std::milli>::zero();
    std::chrono::duration<int64_t, std::milli> computeTime = std::chrono::duration<int64_t, std::milli>::zero();
    uint64_t keyReadTime = 0;
    size_t commBytes = 0;
    printf("Starting training\n");
    
    Dataset d = readDataset(dataset, party);
    int fd;
    fd = openForReading(keyFile + "_" + to_string(0) + "_" + to_string(0) + "_" + std::to_string(0) + ".dat");
    size_t weightsSize, OptimizerSize;

    u64 *mask_W = NULL;
    string wMaskFile = "";
    wMaskFile = weightsDir + "weights_mask_" + std::to_string(party) + "_" + to_string(0) + "_" + to_string(0) + "_" + std::to_string(0) + ".dat";
    mask_W = (u64 *)readFile(wMaskFile, &weightsSize);
    
    u64 *mask_Opt = NULL;
    string OptMaskFile = "";
    OptMaskFile = weightsDir + "optimizer_mask_" + std::to_string(party) + "_" + to_string(0) + "_" + to_string(0) + "_" + std::to_string(0) + ".dat";
    mask_Opt = (u64 *)readFile(OptMaskFile, &OptimizerSize);
    std::cout << "Opt Mask size="  << OptimizerSize << std::endl;
    for (int l = 0; l < epochs; l++)
    {
        for (int k = 0; k < blocks; k++)
        {            
            peer->sync();
            auto startComm = peer->bytesReceived();
            auto start = std::chrono::high_resolution_clock::now();
            lseek(fd, 0, SEEK_SET);
            for (int j = 0; j < blockSz; j++)
            {
                readKey(fd, keySz, curKeyBuf, &keyReadTime);
                peer->sync();
                auto computeStart = std::chrono::high_resolution_clock::now();
                auto labelsIdx = (k * blockSz + j) * batchSz * d.classes;
                int dataIdx = (k * blockSz + j) * d.H * d.W * d.C * batchSz;
                trainModel(m, &curKeyBuf, party, peer, &(d.data[dataIdx]), &(d.labels[labelsIdx]), &g, secfloat, llama, l, l * blocks * blockSz + k * blockSz + j);
                auto computeEnd = std::chrono::high_resolution_clock::now();
                auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(computeEnd - computeStart);
                computeTime += elapsed;  
                curKeyBuf = &keyBuf1[0]; 
            }
            auto end = std::chrono::high_resolution_clock::now();
            auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
            onlineTime += elapsed;
            printf("Online time (ms): %lu\n", elapsed.count());
            auto endComm = peer->bytesReceived();
            commBytes += (endComm - startComm);
            std::pair<double, double> res;
            m->dumpWeights(weightsDir + "masked_weights_" + std::to_string(party) + "_" + std::to_string(l) + "_" + std::to_string(k) + "_" + std::to_string(blockSz-1) + ".dat");
            m->dumpOptimizer(weightsDir + "masked_optimizer_" + std::to_string(party) + "_" + std::to_string(l) + "_" + std::to_string(k) + "_" + std::to_string(blockSz-1) + ".dat", party);
            
            string maskedWFile = weightsDir + "masked_weights_" + std::to_string(party) + "_" + std::to_string(l) + "_" + std::to_string(k) + "_" + std::to_string(blockSz-1) + ".dat";
            std::cout << "===========" << maskedWFile << "==========" << std::endl;
            auto masked_W = (u64 *)readFile(maskedWFile, &weightsSize);
            int N = weightsSize / sizeof(u64);
            auto W = new u64[N];
            
            if (dataset == "mnist")
            {
                std::cout << "Getting loss for MNIST" << std::endl;
                res = getLossMNISTFakeOffline<i64>(modelName, (u64)dcf::orca::global::scale, W, mask_W, masked_W, N);
            }
            else
            {
                std::cout << "Getting loss for CIFAR10" << std::endl;
                res = getLossCIFAR10FakeOffline<i64>(modelName, (u64)dcf::orca::global::scale, W, mask_W, masked_W, N);
            }
            
            for (int i = 0; i < m->layers.size(); i++)
            {
                m->layers[i]->initWeights((u8**)&W, false);
            }
            
            string maskedOptFile = weightsDir + "masked_optimizer_" + std::to_string(party) + "_" + std::to_string(l) + "_" + std::to_string(k) + "_" + std::to_string(blockSz-1) + ".dat";
            std::cout << "===========" << maskedOptFile << "==========" << std::endl;
            auto masked_Opt = (u64 *)readFile(maskedOptFile, &OptimizerSize);
            
            N = OptimizerSize / sizeof(u64);
            auto Opt = new u64[N];
            std::cout << "Weights file=" << maskedOptFile << ", " << OptMaskFile << std::endl;
            std::cout << "Opt Masked Size=" << N << std::endl;
            for (int i = 0; i < N; i++)
            {
                Opt[i] = masked_Opt[i] - (mask_Opt ? mask_Opt[i] : 0);
                cpuMod(Opt[i], dcf::orca::global::bw);
            }
            for (int i = 0; i < m->layers.size(); i++)
            {
                m->layers[i]->initOptimizer((u8**)&Opt, party);
            }
            assert(std::filesystem::remove(maskedWFile));
            assert(std::filesystem::remove(maskedOptFile));
            auto accuracy = res.first;
            auto loss = res.second;
            printf("Accuracy=%lf, Loss=%lf\n", accuracy, loss);
            lossFile << loss << std::endl;
            accFile << accuracy << std::endl;   
        }
    }
    close(fd);


    LlamaConfig::peer->close();
    int iterations = epochs * blocks * blockSz;
    commBytes += inputOnlineComm;
    commBytes += secFloatComm;
    
    // add the wan_time of softmax
    wan_time += numRounds * wanParams.rtt;
    wan_time += (inputOnlineComm + secFloatComm) / (wanParams.comm_bytes_per_ms);
    
    std::ofstream stats(trainingDir + expName + ".txt");
    auto statsString = "Total time taken (ms): " + std::to_string(onlineTime.count()) + "\nTotal bytes communicated: " + std::to_string(commBytes) + "\nSecfloat softmax bytes: " + std::to_string(inputOnlineComm + secFloatComm);
    statsString += "\nWan extra time taken (ms)" + std::to_string(wan_time);
    auto avgKeyReadTime = (double)keyReadTime / (double)iterations;
    auto avgComputeTime = (double)computeTime.count() / (double)iterations;
    double commPerIt = (double)commBytes / (double)iterations;
    auto wan_extra_time = (double)wan_time / (double)iterations;
    statsString += "\nAvg key read time (ms): " + std::to_string(avgKeyReadTime) + "\nAvg compute time (ms): " + std::to_string(avgComputeTime) + "\nAvg wan extra time (ms)" + std::to_string(wan_extra_time);
    statsString += "\nComm per iteration (B): " + std::to_string(commPerIt);
    stats << statsString;
    stats.close();
    std::cout << statsString << std::endl;
    lossFile.close();
    accFile.close();
    destroyCPURandomness();
    destroyGPURandomness();
}

void evaluatorPerf(std::string modelName, std::string dataset, int party, std::string ip, int iterations, int batchSz, int H, int W, int C, bool secfloat, bool momentum, std::string keyDir)
{
    WanParameter wanParams;
    AESGlobalContext g;
    initAESContext(&g);
    initGPUMemPool();
    initGPURandomness();
    initCPURandomness();

    dcf::orca::GPUModel<u64> *m = getGPUModel<u64>(modelName, Tensor<u64>(nullptr, {(u64)batchSz, (u64)H, (u64)W, (u64)C}));
    m->setTrain(momentum);
    size_t inpMemSz = m->inpSz * sizeof(u64);
    auto inp = (u64 *)cpuMalloc(inpMemSz);
    memset(inp, 0, inpMemSz);
    size_t opMemSz = m->batchSz * m->classes * sizeof(u64);
    auto labels = (u64 *)cpuMalloc(opMemSz);
    memset(labels, 0, opMemSz);

    u8 *keyBuf1, *keyBuf2, *curKeyBuf, *nextKeyBuf;
    auto trainingDir = "output/P" + std::to_string(party) + "/training/";
    auto keySzDir = trainingDir + "keysize/";
    u64 keySz = getKeySz(keySzDir, modelName);
    getAlignedBuf(&keyBuf1, keySz);
    getAlignedBuf(&keyBuf2, keySz);
    int curBuf = 0;
    curKeyBuf = keyBuf1;
    nextKeyBuf = keyBuf2;

    SigmaPeer *peer = new GpuPeer(true);
    LlamaBase<u64> *llama = nullptr;

    LlamaConfig::party = party + 2;
    llama = new LlamaBase<u64>();
    if (LlamaConfig::party == SERVER)
        llama->initServer(ip, (char **)&curKeyBuf);
    else
        llama->initClient(ip, (char **)&curKeyBuf);
    peer->peer = LlamaConfig::peer;

    if (secfloat)
        secfloat_init(party + 1, ip);

    std::string keyFile = keyDir + modelName + "_training_key" + std::to_string(party) + ".dat";
    // dropOSPageCache();
    std::chrono::duration<int64_t, std::milli> onlineTime = std::chrono::duration<int64_t, std::milli>::zero();
    std::chrono::duration<int64_t, std::milli> computeTime = std::chrono::duration<int64_t, std::milli>::zero();
    uint64_t keyReadTime = 0;
    size_t commBytes = 0;
    int fd = openForReading(keyFile);
    auto start = std::chrono::high_resolution_clock::now();
    auto startComm = peer->bytesReceived();
    int float_softmax_time = 0;
    for (int j = 0; j < iterations; j++)
    {
        readKey(fd, keySz, curKeyBuf, &keyReadTime);   
        auto computeStart = std::chrono::high_resolution_clock::now();
        trainModelPerf(m, &curKeyBuf, party, peer, inp, labels, &g, secfloat, llama, 0, 0, float_softmax_time);
        auto computeEnd = std::chrono::high_resolution_clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(computeEnd - computeStart);
        computeTime += elapsed;
        curKeyBuf = &keyBuf1[0];
    }
    auto end = std::chrono::high_resolution_clock::now();
    onlineTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    printf("Online time (ms): %lu\n", onlineTime.count());
    auto endComm = peer->bytesReceived();
    commBytes += (endComm - startComm);
    close(fd);

    // add the online comm of softmax
    commBytes += inputOnlineComm;
    commBytes += secFloatComm;
    
    std::cout << "numRounds: " << numRounds << std::endl;
    std::cout << "inputOnlineComm: " << inputOnlineComm << std::endl;
    // add the wan_time of softmax
    wan_time += numRounds * wanParams.rtt;
    wan_time += (inputOnlineComm + secFloatComm) / (wanParams.comm_bytes_per_ms);
    LlamaConfig::peer->close();
    std::ofstream stats(trainingDir + modelName + ".txt");
    auto statsString = "\n" + modelName + "\n";
    statsString += "Total time taken (ms): " + std::to_string(onlineTime.count()) + "\nTotal bytes communicated: " + std::to_string(commBytes) + "\nSecfloat softmax bytes: " + std::to_string(inputOnlineComm + secFloatComm);
    statsString += "\nWan extra time taken (ms)" + std::to_string(wan_time);
    statsString += "\nFLoat SoftMax time take (ms): " + std::to_string(float_softmax_time) + "\n";
    statsString += "\nIterations: " + std::to_string(iterations) + "\n";
    auto totTimeByIt = (double)onlineTime.count() / (double)(iterations - 1);
    auto avgKeyReadTime = (double)keyReadTime / (double)iterations;
    auto avgComputeTIme = (double)computeTime.count() / (double)iterations;
    auto wan_extra_time = (double)wan_time / (double)iterations;
    int truncateComm = 0;
    for (int i = 0; i < m->layers.size(); i++)
    {
        truncateComm += m->layers[i]->s.truncate_comm_bytes;
    }
    double commPerIt = (double)commBytes / (double)iterations;
    statsString += "\nTotal time / iterations (ms): " + std::to_string(totTimeByIt) + "\nAvg key read time (ms): " + std::to_string(avgKeyReadTime) + "\nAvg compute time (ms): " + std::to_string(avgComputeTIme) + "\nAvg wan extra time (ms): " + std::to_string(wan_extra_time);
    statsString += "\nComm per iteration (B): " + std::to_string(commPerIt) + "\n";  
    statsString += "\nTruncate Comm (B): " + std::to_string(truncateComm / 8 * (double)(iterations)) + "\nAvg Truncate Comm (B): " + std::to_string((double)truncateComm / 8);
    stats << statsString;
    stats.close();
    std::cout << statsString << std::endl;
    std::cout << float_softmax_time << std::endl;
    destroyCPURandomness();
    destroyGPURandomness();
}

int global_device = 0;
double wan_time = 0;
int main(int argc, char *argv[])
{
    sytorch_init();
    auto keyDir = std::string(argv[1]);
    auto experiment = std::string(argv[2]);
    auto ip = std::string(argv[3]);
    int party = atoi(argv[4]);
    global_device = atoi(argv[5]);
    using T = u64;
    // Neha: need to fix this later 

    if (experiment.compare("CNN2-FLOAT") == 0)
    {
        int epochs = 2;
        int blocks = 46;
        int blockSz = 10; // 600
        int batchSz = 128;
        evaluatorE2EFakeOffline("CNN2", "mnist", party, ip, "weights/CNN2.dat", false, epochs, blocks, blockSz, batchSz, 28, 28, 1, true, true, keyDir);
    }
    else if (experiment.compare("CNN3-FLOAT") == 0){
        int epochs = 2;
        int blocks = 78;
        int blockSz = 10;
        int batchSz = 64;
        evaluatorE2EFakeOffline("CNN3", "cifar10", party, ip, "weights/CNN3.dat", false, epochs, blocks, blockSz, batchSz, 32, 32, 3, true, true, keyDir);
    }
    else if (experiment.compare("P-SecureML-FLOAT") == 0)
    {
        int epochs = 2;
        int blocks = 46;
        int blockSz = 10;
        int batchSz = 128;
        evaluatorE2EFakeOffline("P-SecureML", "mnist", party, ip, "weights/PSecureMlNoRelu.dat", false, epochs, blocks, blockSz, batchSz, 28, 28, 1, true, true, keyDir);   
    }
    if (experiment.compare("CNN2-COMM") == 0){
        int iterations = 11;
        int batchSz = 128;
        evaluatorPerf("CNN2", "mnist", party, ip, iterations, batchSz, 28, 28, 1, true, true, keyDir);
    }
    else if (experiment.compare("CNN3-COMM") == 0){
        int iterations = 11;
        int batchSz = 64;
        evaluatorPerf("CNN3", "cifar10", party, ip, iterations, batchSz, 32, 32, 3, true, true, keyDir);
    }
    else if (experiment.compare("P-VGG16") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        evaluatorPerf("P-VGG16", "cifar10", party, ip, iterations, batchSz, 32, 32, 3, true, true, keyDir);
    }
    else if (experiment.compare("P-AlexNet") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        evaluatorPerf("P-AlexNet", "cifar10", party, ip, iterations, batchSz, 32, 32, 3, true, true, keyDir);
    }
    else if (experiment.compare("P-LeNet") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        evaluatorPerf("P-LeNet", "mnist", party, ip, iterations, batchSz, 28, 28, 1, true, true, keyDir);
    }
    else if (experiment.compare("P-SecureML") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        evaluatorPerf("P-SecureML", "mnist", party, ip, iterations, batchSz, 28, 28, 1, true, true, keyDir);
    }
    else if (experiment.compare("AlexNet") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        evaluatorPerf("AlexNet", "cifar10", party, ip, iterations, batchSz, 32, 32, 3, true, true, keyDir);
    }
    else if (experiment.compare("Pattern1") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        // 这里设置float softmax是为了更好的减去通信开销，计算pattern
        evaluatorPerf("Pattern1", "mnist", party, ip, iterations, batchSz, 28, 28, 1, true, true, keyDir);
    }
    else if (experiment.compare("Pattern2") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        // 这里设置float softmax是为了更好的减去通信开销，计算pattern
        evaluatorPerf("Pattern2", "mnist", party, ip, iterations, batchSz, 28, 28, 1, true, true, keyDir);
    }
    return 0;
}
