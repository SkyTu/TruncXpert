// 
// Copyright:
// 
// Copyright (c) 2024 Microsoft Research
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <cassert>
#include <cstddef>
#include <cstdint>
#include <chrono>
#include <fcntl.h>
#include <filesystem>
#include <omp.h>
#include <unistd.h>

#include "utils/gpu_data_types.h"
#include "utils/gpu_file_utils.h"
#include "utils/misc_utils.h"
#include "utils/gpu_comms.h"
#include "utils/gpu_mem.h"
#include "utils/hip/hip_runtime_api.h"
#include "utils/gpu_random.h"

#include "cnn_wing.h"

#include <sytorch/backend/llama_base.h>
#include <sytorch/softmax.h>


u64 *gpuGenSoftmaxKey(int batchSz, int numClasses, u64 *d_mask_I, bool secfloat, LlamaBase<u64> *llama)
{
    Tensor4D<u64> inpMask(batchSz, numClasses, 1, 1);
    Tensor4D<u64> softmaxOpMask(batchSz, numClasses, 1, 1);
    size_t memSz = batchSz * numClasses * sizeof(u64);
    moveIntoCPUMem((u8 *)inpMask.data, (u8 *)d_mask_I, memSz, NULL);
    gpuFree(d_mask_I);
    if (secfloat)
    {
        softmax_secfloat(inpMask, softmaxOpMask, wing::global::scale, 1);
        for (int img = 0; img < batchSz; img++)
        {
            for (int c = 0; c < numClasses; c++)
            {
                softmaxOpMask(img, c, 0, 0) = softmaxOpMask(img, c, 0, 0) * (1ULL << wing::global::extra_shift);
            }
        }
    }
    else
    {
        wing_softmax(inpMask, softmaxOpMask, wing::global::scale, wing::global::extra_shift);
    }
    d_mask_I = (u64 *)moveToGPU((u8 *)softmaxOpMask.data, memSz, NULL);
    return d_mask_I;
}

void genModelKey(wing::GPUModel<u64> *m, u8 **bufPtr, int party, AESGlobalContext *g, bool secfloat, LlamaBase<u64> *llama, int epoch)
{
    auto d_mask_I = randomGEOnGpu<u64>(m->inpSz, wing::global::bw);
    auto h_mask_I = (u64*) moveToCPU((u8*)d_mask_I, m->inpSz * sizeof(u64), NULL);
    printf("Generate Model Key\n");
    for (int i = 0; i < 10; i++){
        printf("h_mask_I[%d] = %lu\n", i, h_mask_I[i]);
    }
    u64 *d_mask_O = NULL;
    for (int i = 0; i < m->layers.size(); i++)
    {   
        d_mask_O = m->layers[i]->genForwardKey(bufPtr, party, d_mask_I, g);
        assert(d_mask_O != d_mask_I);
        gpuFree(d_mask_I);
        d_mask_I = d_mask_O;
    }
    d_mask_I = gpuGenSoftmaxKey(m->batchSz, m->classes, d_mask_I, secfloat, llama);
    for (int i = m->layers.size() - 1; i >= 0; i--)
    {
        d_mask_I = m->layers[i]->genBackwardKey(bufPtr, party, d_mask_I, g, epoch);
    }
}

void writeKeySz(std::string dir, std::string modelName, u64 keySz)
{
    makeDir(dir);
    std::ofstream keySzFile(dir + modelName + ".txt");
    keySzFile << keySz;
    keySzFile.close();
}

void dealerE2EFakeOffline(std::string modelName, int party, int epochs, int blocks, int blockSz, int batchSz, int H, int W, int C, bool secfloat, bool momentum, std::string keyDir, int sleepInt, std::string weightsMask = "")
{
    AESGlobalContext g;
    initAESContext(&g);
    initGPURandomness();
    initGPUMemPool();
    sytorch_init();
    // assert(epochs < 6);

    auto expName = modelName + "-" + std::to_string(epochs) + "e-" + std::to_string(blocks) + "b";
    auto trainingDir = "output/P" + std::to_string(party) + "/training/";
    auto lossDir = trainingDir + "loss/" + expName + "/";
    auto keySzDir = trainingDir + "keysize/";
    auto weightsDir = lossDir + "weights/";

    // assumes output/P0/training exists
    makeDir(trainingDir + "loss/");
    makeDir(lossDir);
    makeDir(weightsDir);
    makeDir(keySzDir);

    char one = 1;
    char two = 2;

    std::cout << "before getGPUModel" << std::endl;

    // load the model
    wing::GPUModel<u64> *m = getGPUModel<u64>(modelName, Tensor<u64>(nullptr, {(u64)batchSz, (u64)H, (u64)W, (u64)C}));
    std::cout << "after getGPUModel" << std::endl;
    m->setTrain(momentum);
    m->initWeights(weightsMask, false);

    char *zeros;
    size_t padding, bufSize = 8 * OneGB;
    u8 *startPtr, *curPtr, *tmpPtr1, *tmpPtr2;
    getAlignedBuf(&startPtr, bufSize);

    // initialize llama
    LlamaConfig::party = DEALER;
    auto llama = new LlamaBase<u64>();
    tmpPtr1 = (u8 *)malloc(OneGB);
    bool isServer = party + 2 == SERVER;
    llama->initDealer((char **)(isServer ? &curPtr : &tmpPtr2), (char **)(isServer ? &tmpPtr2 : &curPtr));
    LlamaConfig::bitlength = wing::global::bw;
    std::string keyFile = keyDir + modelName + "_training_key" + std::to_string(party);
    int fd = openForWriting(keyFile + "_" + to_string(0) + "_" + to_string(0) + "_" + std::to_string(0) + ".dat");
    for(int i = 0; i < blockSz; i++){
        curPtr = startPtr;
        tmpPtr2 = tmpPtr1;
        genModelKey(m, &curPtr, party, &g, secfloat, (LlamaBase<u64> *)llama, 0);
        if(i == 0){
            size_t keySz = curPtr - startPtr;
            padding = 4096 - (keySz % 4096);
            keySz += padding;
            zeros = new char[padding];
            memset(zeros, 0, padding);
            writeKeySz(keySzDir, modelName, keySz);
        }
        memcpy(curPtr, zeros, padding);
        curPtr += padding;
        writeKeyBuf(fd, curPtr - startPtr, startPtr);
    }
    m->dumpWeights(weightsDir + "weights_mask_" + std::to_string(party) + "_" + to_string(0) + "_" + to_string(0) + "_" + std::to_string(0) + ".dat");
    m->dumpOptimizerMask(weightsDir + "optimizer_mask_" + std::to_string(party) + "_" + to_string(0) + "_" + to_string(0) + "_" + std::to_string(0) + ".dat", party);
    close(fd);
    delete[] zeros;
    destroyGPURandomness();
}

void dealerE2E(std::string modelName, int party, int epochs, int blocks, int blockSz, int batchSz, int H, int W, int C, bool secfloat, bool momentum, std::string keyDir, int sleepInt, std::string weightsMask = "", bool fake_offline = true)
{
    AESGlobalContext g;
    initAESContext(&g);
    initGPURandomness();
    initGPUMemPool();
    sytorch_init();
    // assert(epochs < 6);

    auto expName = modelName + "-" + std::to_string(epochs) + "e-" + std::to_string(blocks) + "b";
    auto trainingDir = "output/P" + std::to_string(party) + "/training/";
    auto lossDir = trainingDir + "loss/" + expName + "/";
    auto keySzDir = trainingDir + "keysize/";
    auto weightsDir = lossDir + "weights/";

    // assumes output/P0/training exists
    makeDir(trainingDir + "loss/");
    makeDir(lossDir);
    makeDir(weightsDir);
    makeDir(keySzDir);

    char one = 1;
    char two = 2;

    std::cout << "before getGPUModel" << std::endl;

    // load the model
    wing::GPUModel<u64> *m = getGPUModel<u64>(modelName, Tensor<u64>(nullptr, {(u64)batchSz, (u64)H, (u64)W, (u64)C}));
    std::cout << "after getGPUModel" << std::endl;
    m->setTrain(momentum);
    m->initWeights(weightsMask, false);

    char *zeros;
    size_t padding, bufSize = 8 * OneGB;
    u8 *startPtr, *curPtr, *tmpPtr1, *tmpPtr2;
    getAlignedBuf(&startPtr, bufSize);

    // initialize llama
    LlamaConfig::party = DEALER;
    auto llama = new LlamaBase<u64>();
    tmpPtr1 = (u8 *)malloc(OneGB);
    bool isServer = party + 2 == SERVER;
    llama->initDealer((char **)(isServer ? &curPtr : &tmpPtr2), (char **)(isServer ? &tmpPtr2 : &curPtr));
    LlamaConfig::bitlength = wing::global::bw;
    std::string keyFile = keyDir + modelName + "_training_key" + std::to_string(party);
    int fd = openForWriting(keyFile + "_" + to_string(0) + "_" + to_string(0) + "_" + std::to_string(0) + ".dat");
    for (int l = 0; l < epochs; l++)
    {
        for (int k = 0; k < blocks; k++)
        {
            for (int j = 0; j < blockSz; j++)
            {
                printf("Iteration=%u\n", l * blocks * blockSz + k * blockSz + j);
                curPtr = startPtr;
                tmpPtr2 = tmpPtr1;
                genModelKey(m, &curPtr, party, &g, secfloat, (LlamaBase<u64> *)llama, l);
                if (l == 0 && k == 0 && j == 0)
                {
                    size_t keySz = curPtr - startPtr;
                    padding = 4096 - (keySz % 4096);
                    keySz += padding;
                    zeros = new char[padding];
                    memset(zeros, 0, padding);
                    writeKeySz(keySzDir, modelName, keySz);
                }
                memcpy(curPtr, zeros, padding);
                curPtr += padding;
                writeKeyBuf(fd, curPtr - startPtr, startPtr);
            }
            m->dumpWeights(weightsDir + "weights_mask_" + std::to_string(party) + "_" + to_string(l) + "_" + to_string(k) + "_" + std::to_string(blockSz-1) + ".dat");
        }
    }
    close(fd);
    delete[] zeros;
    destroyGPURandomness();
}

void dealerPerf(std::string modelName, int party, int iterations, int batchSz, int H, int W, int C, bool secfloat, bool momentum, std::string keyDir, int sleepInt)
{
    AESGlobalContext g;
    initAESContext(&g);
    initGPURandomness();
    initGPUMemPool();
    sytorch_init();

    auto trainingDir = "output/P" + std::to_string(party) + "/training/";
    auto keySzDir = trainingDir + "keysize/";
    makeDir(keySzDir);

    wing::GPUModel<u64> *m = getGPUModel<u64>(modelName, Tensor<u64>(nullptr, {(u64)batchSz, (u64)H, (u64)W, (u64)C}));
    m->setTrain(momentum);

    char *zeros;
    // Neha: remember to change this later
    size_t padding, bufSize = 23 * OneGB;
    u8 *startPtr, *curPtr, *tmpPtr1, *tmpPtr2;
    getAlignedBuf(&startPtr, bufSize);

    // initialize llama
    LlamaConfig::party = DEALER;
    auto llama = new LlamaBase<u64>();
    tmpPtr1 = (u8 *)malloc(OneGB);
    bool isServer = party + 2 == SERVER;
    llama->initDealer((char **)(isServer ? &curPtr : &tmpPtr2), (char **)(isServer ? &tmpPtr2 : &curPtr));

    std::string keyFile = keyDir + modelName + "_training_key" + std::to_string(party) + ".dat";

    std::cout << keyFile << std::endl;
    int fd = openForWriting(keyFile);

    for (int j = 0; j < iterations; j++)
    {
        curPtr = startPtr;
        tmpPtr2 = tmpPtr1;
        genModelKey(m, &curPtr, party, &g, secfloat, (LlamaBase<u64> *)llama, 0);
        if (j == 0)
        {
            size_t keySz = curPtr - startPtr;
            padding = 4096 - (keySz % 4096);
            zeros = new char[padding];
            memset(zeros, 0, padding);
            keySz += padding;
            writeKeySz(keySzDir, modelName, keySz);
        }
        memcpy(curPtr, zeros, padding);
        curPtr += padding;
        writeKeyBuf(fd, curPtr - startPtr, startPtr);
    }
    assert(0 == fsync(fd) && "sync error!");
    close(fd);
    printf("Sleeping for %d seconds.\n", sleepInt);
    delete[] zeros;
    destroyGPURandomness();
}

int global_device = 0;

int main(int argc, char *argv[])
{
    int party = atoi(argv[1]);
    auto keyDir = std::string(argv[2]);
    auto experiment = std::string(argv[3]);
    global_device = atoi(argv[4]);

    omp_set_num_threads(32);
    if (experiment.compare("CNN2") == 0){
        int epochs = 2;
        int blocks = 46;
        int blockSz = 10;
        int batchSz = 128;
        dealerE2EFakeOffline("CNN2", party, epochs, blocks, blockSz, batchSz, 28, 28, 1, false, true, keyDir, 300, "");
        // dealerE2E("CNN2", party, epochs, blocks, blockSz, batchSz, 28, 28, 1, false, true, keyDir, 300, "");
    }
    else if (experiment.compare("CNN3-FLOAT") == 0){
        int epochs = 2;
        int blocks = 78;
        int blockSz = 10;
        int batchSz = 64;
        dealerE2EFakeOffline("CNN3", party, epochs, blocks, blockSz, batchSz, 32, 32, 3, true, true, keyDir, 300, "");
    }
    else if (experiment.compare("CNN3") == 0){
        int epochs = 2;
        int blocks = 78;
        int blockSz = 10;
        int batchSz = 64;
        dealerE2EFakeOffline("CNN3", party, epochs, blocks, blockSz, batchSz, 32, 32, 3, false, true, keyDir, 300, "");
    }
    else if (experiment.compare("P-SecureML-Train") == 0)
    {
        int epochs = 2;
        int blocks = 46;
        int blockSz = 10;
        int batchSz = 128;
        dealerE2EFakeOffline("P-SecureML", party, epochs, blocks, blockSz, batchSz, 28, 28, 1, false, true, keyDir, 300, "");   
    }
    if (experiment.compare("CNN2-COMM") == 0){
        int iterations = 11;
        int batchSz = 128;
        dealerPerf("CNN2", party, iterations, batchSz, 28, 28, 1, false, true, keyDir, 300);
    }
    else if (experiment.compare("CNN3-COMM") == 0){
        int iterations = 11;
        int batchSz = 64;
        dealerPerf("CNN3", party, iterations, batchSz, 32, 32, 3, false, true, keyDir, 300);
    }
    else if (experiment.compare("P-VGG16") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        dealerPerf("P-VGG16", party, iterations, batchSz, 32, 32, 3, false, true, keyDir, 300);
    }
    else if (experiment.compare("P-AlexNet") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        dealerPerf("P-AlexNet", party, iterations, batchSz, 32, 32, 3, false, true, keyDir, 300);
    }
    else if (experiment.compare("P-LeNet") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        dealerPerf("P-LeNet", party, iterations, batchSz, 28, 28, 1, false, true, keyDir, 60);
    }
    else if (experiment.compare("P-SecureML") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        dealerPerf("P-SecureML", party, iterations, batchSz, 28, 28, 1, false, true, keyDir, 60);
    }
    else if (experiment.compare("AlexNet") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        dealerPerf("AlexNet", party, iterations, batchSz, 32, 32, 3, false, true, keyDir, 300);
    }
    else if (experiment.compare("Pattern1") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        // 这里设置float softmax是为了更好的减去通信开销，计算pattern
        dealerPerf("Pattern1", party, iterations, batchSz, 28, 28, 1, true, true, keyDir, 300);
    }
    else if (experiment.compare("Pattern2") == 0)
    {
        int iterations = 11;
        int batchSz = 128;
        // 这里设置float softmax是为了更好的减去通信开销，计算pattern
        dealerPerf("Pattern2", party, iterations, batchSz, 28, 28, 1, true, true, keyDir, 300);
    }
    return 0;
}
