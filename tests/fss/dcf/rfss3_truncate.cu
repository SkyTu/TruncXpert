// Author: Neha Jawalkar
// Copyright:
// 
// Copyright (c) 2024 Microsoft Research
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <stdio.h>
#include <cassert>
#include <cstdint>

#include "utils/gpu_file_utils.h"
#include "utils/misc_utils.h"
#include "utils/gpu_comms.h"
#include "utils/gpu_mem.h"
#include "utils/gpu_random.h"

#include "fss/dcf/gpu_truncate.h"

using T = u64;

inline T cpuMsb(T x, int bin){
    return ((x >> (bin - 1)) & T(1));
}

int main(int argc, char *argv[]) {
    AESGlobalContext g;
    initAESContext(&g);
    initGPURandomness();
    // initCommBufs(true);
    int bin = 64;
    int bout = 64;
    int shift = 16;
    int N = atoi(argv[3]);
    int party = atoi(argv[1]);
    
    auto peer = new GpuPeer(false);
    peer->connect(party, argv[2]);

    // generate rin
    auto h_X = new T[N];
    auto d_X_0 = randomGEOnGpu<T>(N, bin);
    auto d_X_1 = randomGEOnGpu<T>(N, bin);
    auto d_X = (T *)gpuMalloc(N * sizeof(T));
    auto d_mask = (T *)gpuMalloc(N * sizeof(T));
    hipError_t err = hipMemset(d_mask, 0, N * sizeof(T));
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    gpuLinearComb(64, N, d_X, T(1), d_X_0, T(1), d_X_1);
    h_X = (T *)moveToCPU((u8 *)d_X, N * sizeof(T), NULL);    
    int bw = 64;

    u8 *startPtr, *curPtr;
    size_t keyBufSz = 10 * OneGB;
    getKeyBuf(&startPtr, &curPtr, keyBufSz);
    T* h_r = (T*) cpuMalloc(N * sizeof(T));
    dcf::TruncateType t = dcf::TruncateType::StochasticTR;

    // generate TReKey
    auto d_truncateMask = dcf::genGPUTReKey(&curPtr, party, bin, bin-shift, shift, N, d_mask, &g, h_r);
    assert(curPtr - startPtr < keyBufSz);
    auto h_truncateMask = (T*) moveToCPU((u8*) d_truncateMask, N * sizeof(T), NULL);

    
    curPtr = startPtr;
    std::cout << "Reading key\n";
    auto k = dcf::readGPUStTRKey<T>(&curPtr);
    auto h_TRe = new T[N];
    if(party == 1){
        dcf::gpuTRe(k, party, peer, d_X_1, &g, (Stats*) NULL);
        h_TRe = (T*) moveToCPU((u8*) d_X_1, N * sizeof(T), NULL);
    }
    else{
        dcf::gpuTRe(k, party, peer, d_X_0, &g, (Stats*) NULL);
        h_TRe = (T*) moveToCPU((u8*) d_X_0, N * sizeof(T), NULL);
    }
    // dcf::gpuZeroExt(k.ZeroExtKey, party, peer, d_masked_X, &g, (Stats*) NULL);
    // auto h_ZeroExt = (T*) moveToCPU((u8*) d_masked_X, N * sizeof(T), NULL);
    // 计算结果是存在d_mask_X的
    destroyGPURandomness();

    for (int i = 0; i < N; i++)
    {
        auto unmasked_TRe = h_TRe[i];
        auto o = cpuArs(h_X[i], bin, shift);
        cpuMod(o, bin-shift);
        if (o != unmasked_TRe)
            printf("%d: h_x = %ld, real_truncate = %ld, stTR_res = %ld\n", i, h_X[i], o, unmasked_TRe);
    }
    std::cout << peer->peer->keyBuf->bytesSent << std::endl;
}