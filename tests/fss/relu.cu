#include "hip/hip_runtime.h"
// Author: Neha Jawalkar
// Copyright:
// 
// Copyright (c) 2024 Microsoft Research
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "utils/gpu_data_types.h"
#include "utils/gpu_file_utils.h"
#include "utils/misc_utils.h"
#include "utils/gpu_mem.h"
#include "utils/gpu_random.h"
#include "utils/gpu_comms.h"

#include "fss/dcf/gpu_relu.h"

#include <cassert>
#include <sytorch/tensor.h>

using T = u64;

int main(int argc, char *argv[])
{
    // initCommBufs(true);
    initGPUMemPool();
    AESGlobalContext g;
    initAESContext(&g);
    int bw = 64;
    int N = atoi(argv[3]); //8;
    // const u64 p = (1ULL << 16) - 1;
    const u64 p = 0;
    printf("\n p is %ld", p);
    const u64 q = p;
    int party = atoi(argv[1]);
    printf("\n party is %d", party);
    auto peer = new GpuPeer(true);
    peer->connect(party, argv[2]);
    printf("\n connected!");
    uint8_t *startPtr, *curPtr;
    printf("\n OneGB is %d\n", OneGB);
    getKeyBuf(&startPtr, &curPtr, 4 * OneGB);

    initGPURandomness();
    // d_mask_X 是输入的掩码
    auto d_mask_X = randomGEOnGpu<T>(N, bw);
    // h_mask_X 是在CPU上输入的掩码
    auto h_mask_X = (T *)moveToCPU((u8 *)d_mask_X, N * sizeof(T), NULL);
    T *h_X;
    // d_masked_X 是masked的输入
    auto d_masked_X = getMaskedInputOnGpu(N, bw, d_mask_X, &h_X);

    auto d_temp = dcf::gpuGenTwoRoundReluKey(&curPtr, party, bw, bw, N, d_mask_X, &g);
    printf("Key size=%lu\n", curPtr - startPtr);

    auto d_dreluMask = d_temp.first;
    auto d_reluMask = d_temp.second;

    auto h_mask_O = (T *)moveToCPU((u8 *)d_reluMask, N * sizeof(T), NULL);
    auto k1 = dcf::readTwoRoundReluKey<T>(&startPtr);
    T *d_O;
    for (int i = 0; i < 10; i++)
    {
        peer->sync();
        auto start = std::chrono::high_resolution_clock::now();
        auto d_temp = dcf::gpuTwoRoundRelu(peer, party, k1, d_masked_X, &g, (Stats *)NULL);
        auto d_drelu = d_temp.first;
        gpuFree(d_drelu);
        d_O = d_temp.second;
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed = end - start;
        printf("Time taken=%lu micros\n", std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count());
    }

    auto h_O = (T *)moveToCPU((uint8_t *)d_O, N * sizeof(T), (Stats *)NULL);
    gpuFree(d_O);
    destroyGPURandomness();

    printf("bw=%d, bw=%d, N=%d\n", bw, bw, N);
    for (int i = 0; i < N; i++)
    {
        auto unmasked_O = (h_O[i] - h_mask_O[i]);
        cpuMod(unmasked_O, bw);
        auto o = h_X[i] * (1 - (h_X[i] >> (bw - 1)));
        cpuMod(o, bw);
        // auto o = std::min(std::abs((i64) h_X[i]), (i64) max);
        if (i < 10)
            printf("%d: %ld, %ld, %ld, %ld\n", i, h_X[i], o, unmasked_O, h_mask_X[i]);
        assert(o == unmasked_O);
    }

    return 0;
}